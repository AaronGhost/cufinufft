#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>

#include <cufinufft.h>
#include "../cuspreadinterp.h"
#include "../cudeconvolve.h"
#include "../memtransfer.h"

using namespace std;

int cufinufft3d1_exec(CUCPX* d_c, CUCPX* d_fk, cufinufft_plan *d_plan)
/*  
	3D Type-1 NUFFT

	This function is called in "exec" stage (See ../cufinufft.cu).
	It includes (copied from doc in finufft library)
		Step 1: spread data to oversampled regular mesh using kernel
		Step 2: compute FFT on uniform mesh
		Step 3: deconvolve by division of each Fourier mode independently by the
		        Fourier series coefficient of the kernel.

	Melody Shih 07/25/19		
*/
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	int blksize; 
	int ier;
	CUCPX* d_fkstart;
	CUCPX* d_cstart;
	for(int i=0; i*d_plan->maxbatchsize < d_plan->ntransf; i++){
		blksize = min(d_plan->ntransf - i*d_plan->maxbatchsize, 
			d_plan->maxbatchsize);
		d_cstart = d_c + i*d_plan->maxbatchsize*d_plan->M;
		d_fkstart = d_fk + i*d_plan->maxbatchsize*d_plan->ms*d_plan->mt*
			d_plan->mu;

		d_plan->c = d_cstart;
		d_plan->fk = d_fkstart;

		checkCudaErrors(hipMemset(d_plan->fw,0,d_plan->maxbatchsize*
					d_plan->nf1*d_plan->nf2*d_plan->nf3*sizeof(CUCPX)));
#ifdef TIME
		float milliseconds = 0;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tInitialize fw\t\t %.3g s\n", milliseconds/1000);
#endif
		// Step 1: Spread
		hipEventRecord(start);
		ier = cuspread3d(d_plan, blksize);
		if(ier != 0 ){
			printf("error: cuspread3d, method(%d)\n", d_plan->opts.gpu_method);
			return 0;
		}
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tSpread (%d)\t\t %.3g s\n", milliseconds/1000, 
			d_plan->opts.gpu_method);
#endif
		// Step 2: FFT
		hipEventRecord(start);
		CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds/1000);
#endif

		// Step 3: deconvolve and shuffle
		hipEventRecord(start);
		cudeconvolve3d(d_plan, blksize);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tDeconvolve\t\t %.3g s\n", milliseconds/1000);
#endif
	}
	return ier;
}

int cufinufft3d2_exec(CUCPX* d_c, CUCPX* d_fk, cufinufft_plan *d_plan)
/*  
	3D Type-2 NUFFT

	This function is called in "exec" stage (See ../cufinufft.cu).
	It includes (copied from doc in finufft library)
		Step 1: deconvolve (amplify) each Fourier mode, dividing by kernel 
		        Fourier coeff
		Step 2: compute FFT on uniform mesh
		Step 3: interpolate data to regular mesh

	Melody Shih 07/25/19		
*/
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int blksize;
	int ier;
	CUCPX* d_fkstart;
	CUCPX* d_cstart;
	for(int i=0; i*d_plan->maxbatchsize < d_plan->ntransf; i++){
		blksize = min(d_plan->ntransf - i*d_plan->maxbatchsize, 
			d_plan->maxbatchsize);
		d_cstart  = d_c  + i*d_plan->maxbatchsize*d_plan->M;
		d_fkstart = d_fk + i*d_plan->maxbatchsize*d_plan->ms*d_plan->mt*
			d_plan->mu;

		d_plan->c = d_cstart;
		d_plan->fk = d_fkstart;

		// Step 1: amplify Fourier coeffs fk and copy into upsampled array fw
		hipEventRecord(start);
		cudeconvolve3d(d_plan, blksize);
#ifdef TIME
		float milliseconds = 0;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tAmplify & Copy fktofw\t %.3g s\n", milliseconds/1000);
#endif
		// Step 2: FFT
		hipEventRecord(start);
		hipDeviceSynchronize();
		CUFFT_EX(d_plan->fftplan, d_plan->fw, d_plan->fw, d_plan->iflag);
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tCUFFT Exec\t\t %.3g s\n", milliseconds/1000);
#endif

		// Step 3: deconvolve and shuffle
		hipEventRecord(start);
		ier = cuinterp3d(d_plan, blksize);
		if(ier != 0 ){
			printf("error: cuinterp3d, method(%d)\n", d_plan->opts.gpu_method);
			return 0;
		}
#ifdef TIME
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("[time  ] \tUnspread (%d)\t\t %.3g s\n", milliseconds/1000,
			d_plan->opts.gpu_method);
#endif

		hipEventRecord(start);
#if 0
		if(d_plan->nstreams != 1)
			hipDeviceSynchronize();
#endif
	}
	return ier;
}
